#include "hip/hip_runtime.h"

#include "common.hpp"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define N (1024 * 512)

#ifndef KERNEL
# error "kernel not defined"
#endif

__global__ void good(int *a, int *b, int *c)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   c[i] = a[i] * b[i];
}

__global__ void bad(int *a, int *b, int *c)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   i = (i * i) % N;

   c[i] = a[i] * b[i];
}

int main()
{
    int a[N] = {};
    int b[N] = {};
    int c[N] = {};

    int *ca, *cb, *cc;

    for (int i = 0; i < N; i++) {
        a[i] = i + 1;
        b[i] = i * i;
    }

    check(hipSetDevice(0), "set device fail");

    check(hipMalloc(&ca, N * sizeof(int)), "malloc fail");
    check(hipMalloc(&cb, N * sizeof(int)), "malloc fail");
    check(hipMalloc(&cc, N * sizeof(int)), "malloc fail");

    check(hipMemcpy(ca, a, N * sizeof(int), hipMemcpyHostToDevice), "memcpy fail");
    check(hipMemcpy(cb, b, N * sizeof(int), hipMemcpyHostToDevice), "memcpy fail");

    timer_start();
    KERNEL<<<N / 32, 32>>> (ca, cb, cc);
    float t = timer_stop();

    check(hipMemcpy(c, cc, N * sizeof(int), hipMemcpyDeviceToHost), "memcpy fail");

    check(hipFree(ca), "free fail");
    check(hipFree(cb), "free fail");
    check(hipFree(cc), "free fail");

    printf("exec time: %g\n", t);
    for (int i = 0; i < min(N, 20); i++) {
        printf("%d: %d %d %d\n", i, a[i], b[i], c[i]);
    }
    std::cout << "execution time: " << t << '\n';
}