#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "common.hpp"

__global__ void dummyKernel()
{
    __shared__ int buff[32];
    buff[threadIdx.x] = 0;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    buff[threadIdx.x] += idx;
}

int main()
{
   int device;
   check(hipGetDevice(&device), "get device");

   int maxThreadsPerBlock;
   check(hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, device), "get attribute");
   std::cout << "max threads per block: " << maxThreadsPerBlock << '\n';

   int blockSize = 256;
   int minGridSize;
   int occupancy;

   check(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, dummyKernel, 0, 0), "occupancy block size");
   std::cout << "min grid size: " << minGridSize << '\n';
   std::cout << "optimal block size: " << blockSize << '\n';

   int smCount;
   check(hipOccupancyMaxActiveBlocksPerMultiprocessor(&occupancy, dummyKernel, blockSize, 0), "occupancy active blocks");
   check(hipDeviceGetAttribute(&smCount, hipDeviceAttributeMultiprocessorCount, device), "get attribute");

   std::cout << "max active blocks: " << occupancy * smCount << std::endl;

    return 0;
}
