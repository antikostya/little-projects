#include "hip/hip_runtime.h"

#include "common.hpp"

#include <stdio.h>
#include <iostream>
#include <stdlib.h>

#define N (512)

__global__ void kernel (float *a, float *b, float *c)
{
   long idx = threadIdx.x + blockIdx.x * blockDim.x;
   c[idx] = a[idx] + b[idx];
}

int main()
{
    float a[N] = {};
    float b[N] = {};
    float c[N] = {};

    float *ca, *cb, *cc;

    for (long i = 0; i < N; i++) {
        a[i] = i + 1;
        b[i] = i * i;
    }

    check(hipSetDevice(0), "set device fail");

    check(hipMalloc(&ca, N * sizeof(float)), "malloc fail");
    check(hipMalloc(&cb, N * sizeof(float)), "malloc fail");
    check(hipMalloc(&cc, N * sizeof(float)), "malloc fail");

    std::cout << "alloc 1\n";
    check(hipMemcpy(ca, a, N * sizeof(float), hipMemcpyHostToDevice), "memcpy fail");
    std::cout << "alloc 2\n";
    check(hipMemcpy(cb, b, N * sizeof(float), hipMemcpyHostToDevice), "memcpy fail");

    timer_start();
    std::cout << "running\n";
    kernel<<<dim3(1, 1, 1), dim3(512, 1, 1)>>> (ca, cb, cc);
    float gpuTime = timer_stop();

    check(hipMemcpy(c, cc, N * sizeof(float), hipMemcpyDeviceToHost), "memcpy fail");

    check(hipFree(ca), "free fail");
    check(hipFree(cb), "free fail");
    check(hipFree(cc), "free fail");

    for (long i = 0; i < N; i++) {
        if (i < 10 || i > N - 10) {
            printf("%f %f %f\n", a[i], b[i], c[i]);
        }
    }
    
    std::cout << "execution time: " << gpuTime << '\n';
}